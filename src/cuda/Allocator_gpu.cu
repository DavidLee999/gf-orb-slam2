/**
* This file is part of GF-ORB-SLAM2.
*
* Copyright (C) 2019 Yipu Zhao <yipu dot zhao at gatech dot edu> 
* (Georgia Institute of Technology)
* For more information see 
* <https://sites.google.com/site/zhaoyipu/good-feature-visual-slam>
*
* GF-ORB-SLAM2 is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* GF-ORB-SLAM is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with GF-ORB-SLAM2. If not, see <http://www.gnu.org/licenses/>.
*/

#include <cassert>
#include <hip/hip_runtime_api.h>
#include <cuda/Allocator.hpp>

namespace ORB_SLAM2 { namespace cuda {

size_t Allocator::getPitch(size_t widthSize){
    return 128 + widthSize - widthSize%128;
}

bool Allocator::allocate(cv::cuda::GpuMat* mat, int rows, int cols, size_t elemSize)
{
    if (rows > 1 && cols > 1)
    {
        mat->step = getPitch(elemSize * cols);
        checkCudaErrors(hipMallocManaged(&mat->data, mat->step * rows));
    }
    else
    {
        // Single row or single column must be continuous
        checkCudaErrors(hipMallocManaged(&mat->data, elemSize * cols * rows));
        mat->step = elemSize * cols;
    }

    mat->refcount = (int*) new int();

    return true;
}

void Allocator::free(cv::cuda::GpuMat* mat)
{
    checkCudaErrors(hipFree(mat->datastart));
    delete mat->refcount;
}

cv::cuda::GpuMat::Allocator * gpu_mat_allocator;

} }


namespace {
  using namespace ORB_SLAM2;

  void __attribute__((constructor)) init() {
    // Setup GPU Memory Management
    cuda::gpu_mat_allocator = new cuda::Allocator();
    // cv::cuda::GpuMat::setDefaultAllocator(cuda::gpu_mat_allocator);
  }
}
